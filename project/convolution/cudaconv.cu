#include "hip/hip_runtime.h"
#include <iostream>
#define STB_IMAGE_IMPLEMENTATION
#include "../stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../stb/stb_image_write.h"

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 32
#define PI 3.14159265358979323846
#define WINDOW 6

__global__
void conv2(float *A, float *B,uint8_t *C, int height, int width, int window){
	int row = threadIdx.x + blockIdx.x*blockDim.x;
	int col = threadIdx.y + blockIdx.y*blockDim.y;
	int i = row + col*width;
	float sum = 0;
	if(i<height*width){
		for(int p = -window; p <= window; ++p){     //loop over window
			for(int q = -window; q <= window; ++q){
				int pixel = i + p*width + q;
				if((pixel < 0) | (pixel >= height*width))  //if outside of vertical bound, continue
					continue;
				int temp = i % width;
				if((temp + q < 0) | (temp + q >= width)) //if outside of horiz bound, continue
					continue;
				sum += A[pixel]*B[(p + window)*(2*window + 1)+(q + window)];
			}
		}
		C[i] = (uint8_t)((int)sum);
	}
}

int main(int argc, char** argv) {
    	//Initiialize variables for image
    	int width, height, bpp;
	//initialize timing variables and create events
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    	//load in image
	uint8_t* img = stbi_load("../images/imageBig.jpg", &width, &height, &bpp, 3);
    	if(img == NULL) {
          	printf("Error in loading the image\n");
         	exit(1);
    	}
    	//convert to grayscale and free image
    	float* grayimg = new float[width*height];
    	for(int i = 0; i < width*height; ++i)
        	grayimg[i] = (.3*(int)img[3*i] + .59*(int)img[3*i+1]+.11*(int)img[3*i+2]);
    	stbi_image_free(img);

	//initialize device copy of grayscale image, allocate it, and copy to device
	float *d_gray;
	hipMalloc((void**)&d_gray,height*width*sizeof(float));
	hipMemcpy(d_gray,grayimg,height*width*sizeof(float),hipMemcpyHostToDevice);

    	//load in scale-space kernel
    	    //Initialize and define scale-space kernel
    float t = 1.0;
    int winsize = (2*WINDOW+1)*(2*WINDOW+1);
    float* scalekernel = new float[winsize];
    for (int i = -WINDOW; i <= WINDOW; ++i){
        for(int j = -WINDOW; j <= WINDOW; ++j){
            scalekernel[j+WINDOW + (i+WINDOW)*(2*WINDOW+1)] = exp(-(i*i+j*j)/(2*t))/(2*PI*t);
        }
    }

	//initialize device copy of scale-space kernel, allocate space, and copy to device
	float *d_scalekernel;
	hipMalloc((void**)&d_scalekernel,winsize*sizeof(float));
	hipMemcpy(d_scalekernel,scalekernel,winsize*sizeof(float),hipMemcpyHostToDevice);

	//initialize image to store result, initialize variable for device, and allocate space in device
	uint8_t* finalimg = new uint8_t[width*height];
	uint8_t* d_finalimg;
	hipMalloc((void**)&d_finalimg,width*height*sizeof(uint8_t));

	//initialize blocksize and gridsize
	dim3 dimBlock(BLOCK_SIZE_X,BLOCK_SIZE_Y);
	dim3 dimGrid(ceil(width/(float)dimBlock.x),ceil(height/(float)dimBlock.y));

	// Loop over image pixels
	hipEventRecord(start);
	conv2<<<dimGrid,dimBlock>>>(d_gray, d_scalekernel, d_finalimg, height, width, WINDOW);
	hipEventRecord(stop);
    	printf("Done!\n");

	//copy result to host
	hipMemcpy(finalimg,d_finalimg,height*width*sizeof(uint8_t),hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);

	//save image!
	int channels = 1;
    	stbi_write_jpg("../images/convcuda.jpg", width, height, channels, finalimg, 100);

	float milliseconds;
	hipEventElapsedTime(&milliseconds,start,stop);
	std::cout<< milliseconds/1000.0 << '\n';
	//free device variables
	hipFree(d_gray);
	hipFree(d_scalekernel);
	hipFree(d_finalimg);

	//free host variables
    	delete [] grayimg;
	delete [] scalekernel;
    	delete [] finalimg;

	return 0;
}

